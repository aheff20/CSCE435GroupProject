#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include "../../Utils/helper_functions.h"

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* data_init = "data_init";
const char* comm = "comm";
const char* comm_large = "comm_large";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* correctness_check = "correctness_check";

// CUDA kernel function for bubble sort step
__global__ void bubble_sort_step(float *dev_values, int size, bool even_phase) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = 2 * idx + (even_phase ? 0 : 1); 

    if (even_phase) {
        // Even phase: Compare elements at even index with the next element
        if (i < size - 1 - (size % 2) && dev_values[i] > dev_values[i + 1]) {
            float temp = dev_values[i];
            dev_values[i] = dev_values[i + 1];
            dev_values[i + 1] = temp;
        }
    } else {
        // Odd phase: Compare elements at odd index with the next element
        if (i < size - 1 && dev_values[i] > dev_values[i + 1]) {
            float temp = dev_values[i];
            dev_values[i] = dev_values[i + 1];
            dev_values[i + 1] = temp;
        }
    }
}

// Host function to sort an array using bubble sort on the GPU
void bubbleSort(float *values, int size, int *kernel_calls) {
    float *dev_values;
    hipMalloc((void**)&dev_values, size * sizeof(float));
    size_t bytes = size * sizeof(float);

    // Copy data from host to device
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    hipMemcpy(dev_values, values, bytes, hipMemcpyHostToDevice);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    int threads = THREADS;
    int blocks = (size + threads - 1) / threads;

    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    
    for (int i = 0; i < size; ++i) {
        bool even_phase = (i % 2) == 0;
        bubble_sort_step<<<blocks, threads>>>(dev_values, size, even_phase);
        hipDeviceSynchronize();

        (*kernel_calls)++;
    }
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    // Copy the sorted array back to the host
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    hipMemcpy(values, dev_values, bytes, hipMemcpyDeviceToHost);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    // Cleanup
    hipFree(dev_values);
}

int main(int argc, char *argv[]) {
    CALI_CXX_MARK_FUNCTION;

    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads per block: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    float *values = (float*)malloc(NUM_VALS * sizeof(float));
    // CALI_CXX_MARK_FUNCTION;

    // Initialize data
    CALI_MARK_BEGIN("data_init");
    array_fill_random(values, NUM_VALS);
    CALI_MARK_END("data_init");

    // Declare variables for timing information
    int kernel_calls = 0;

    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    bubbleSort(values, NUM_VALS, &kernel_calls);
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    CALI_MARK_BEGIN(correctness_check);

    bool correct = check_sorted(values, NUM_VALS);
    if (correct){
        printf("Array was sorted correctly!");
    }
    else{
         printf("Array was incorrectly sorted!");
    }
    
    CALI_MARK_END(correctness_check);

    // Output timing information
    printf("Total Kernel Calls: %d\n", kernel_calls);

    // Deallocate memory
    free(values);

    const char* algorithm = "Bubble sort";
    const char* programmingModel = "CUDA";
    const char* datatype = "Float";
    int sizeOfDatatype = sizeof(float);
    int inputSize = NUM_VALS;
    const char* inputType = "Random";
    int num_procs = 1;
    int num_threads = THREADS;
    int num_blocks = BLOCKS;
    int group_number = 1;
    const char* implementation_source = "Online/AI";

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", algorithm); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", programmingModel); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", datatype); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeOfDatatype); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", inputSize); // The number of elements in input dataset (1000)
    adiak::value("InputType", inputType); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_procs", num_procs); // The number of processors (MPI ranks)
    adiak::value("num_threads", num_threads); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", num_blocks); // The number of CUDA blocks 
    adiak::value("group_num", group_number); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", implementation_source); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten")

    // Finalize and clean up
    adiak::fini();

    return 0;
}
